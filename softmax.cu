
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cfloat>

/*
    General algorithm
    -   1. Finding max using reduction and shared memory
    -   2. Finding exp and sum using reduction and shared memory
    -   3. compute the softmax per thread. 
*/
__global__
void softmax(float *in, float *out, int h, int w){
    __shared__ float tile[1024];
    int x = blockIdx.x;
    int y = threadIdx.x;

    /* computing max using reduction and shared memory */
    float max_v = -FLT_MAX;
    for(int i = y; i < w; i += blockDim.x){
        max_v = fmaxf(max_v, in[x*w + i]);
    }
    tile[y] = max_v;
    __syncthreads();
    for(int stride = blockDim.x/2; stride > 0; stride /= 2){
        if(threadIdx.x < stride){
            tile[threadIdx.x] = fmaxf(tile[threadIdx.x], tile[threadIdx.x + stride]);
        }
        __syncthreads();
    }
    max_v = tile[0];

    /* computing exp and sum using reduction and shared memory */
    float sum = 0.0f;
    for (int i = y; i < w; i += blockDim.x) {
        out[x*w + i] = expf(in[x*w + i] - max_v);
        sum += out[x*w + i];
    }
    tile[y] = sum; 
    __syncthreads();
    for(int stride = blockDim.x/2; stride > 0; stride /= 2){
        if(threadIdx.x < stride){
            tile[threadIdx.x] += tile[threadIdx.x + stride];
        }
        __syncthreads();
    }
    sum = tile[0];
    __syncthreads();

    for(int i=y; i < w; i+= blockDim.x){
        out[x*w + i] /= sum;
    }

    return;
}

extern "C" void launch_softmax(float *in, float *out, int h, int w) {
    int block_size_x = std::min(1024, w);
    dim3 block_size(block_size_x);
    dim3 grid_size(h);
    softmax<<<grid_size, block_size>>>(in, out, h, w);
}